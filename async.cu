

#include <hip/hip_runtime.h>
#include <stdio.h>

// Convenience function for checking CUDA runtime API results
// can be wrapped around any runtime API call. No-op in release builds.
inline
hipError_t GCE(hipError_t result)
{
#if defined(DEBUG) || defined(_DEBUG)
  if (result != cudaSuccess) {
    fprintf(stderr, "CUDA Runtime Error: %s\n", cudaGetErrorString(result));
    assert(result == cudaSuccess);
  }
#endif
  return result;
}

__global__ void kernel(float *a, int offset) {
  int i = offset + threadIdx.x + blockIdx.x*blockDim.x;
  float x = (float)i;
  float s = sinf(x); 
  float c = cosf(x);
  a[i] = a[i] + sqrtf(s*s+c*c);
}

float maxError(float *a, int n) {
  float maxE = 0;
  for (int i = 0; i < n; i++) {
    float error = fabs(a[i]-1.0f);
    if (error > maxE) maxE = error;
  }
  return maxE;
}

int main(int argc, char **argv)
{
  const int blockSize = 256, nStreams = 4;
  
  const int n = 4 * 4096 * blockSize * nStreams;
  
  const int streamSize = n / nStreams;
  const int streamBytes = streamSize * sizeof(float);
  const int bytes = n * sizeof(float);
   
  float *a, *d_a;
  GCE( hipHostMalloc((void**)&a,   bytes, hipHostMallocDefault) );      // host pinned
  GCE(     hipMalloc((void**)&d_a, bytes) );      // device

  float ms; // elapsed time in milliseconds
  
  // create events and streams
  hipEvent_t startEvent, stopEvent;
  hipStream_t stream[nStreams];
  GCE( hipEventCreate(&startEvent) );
  GCE( hipEventCreate(&stopEvent) );
  
  for (int i = 0; i < nStreams; ++i)
    GCE( hipStreamCreate(&stream[i]) );
  
  // baseline case - sequential transfer and execute
  memset(a, 0, bytes);
  
  GCE( hipEventRecord(startEvent,0) );
  GCE( hipMemcpy(d_a, a, bytes, hipMemcpyHostToDevice) );

  kernel<<<n/blockSize, blockSize>>>(d_a, 0);

  GCE( hipMemcpy(a, d_a, bytes, hipMemcpyDeviceToHost) );
  GCE( hipEventRecord(stopEvent, 0) );
  GCE( hipEventSynchronize(stopEvent) );
  GCE( hipEventElapsedTime(&ms, startEvent, stopEvent) );

  printf("Time for sequential transfer and execute (ms): %f\n", ms);
  printf("  max error: %e\n", maxError(a, n));

  // asynchronous version 1: loop over {copy, kernel, copy}
  memset(a, 0, bytes);
  GCE( hipEventRecord(startEvent,0) );
  for (int i = 0; i < nStreams; ++i) {
    int offset = i * streamSize;
    GCE( hipMemcpyAsync(&d_a[offset], &a[offset], 
                               streamBytes, hipMemcpyHostToDevice, 
                               stream[i]) );
    kernel<<<streamSize/blockSize, blockSize, 0, stream[i]>>>(d_a, offset);
    GCE( hipMemcpyAsync(&a[offset], &d_a[offset], 
                               streamBytes, hipMemcpyDeviceToHost,
                               stream[i]) );
  }
  GCE( hipEventRecord(stopEvent, 0) );
  GCE( hipEventSynchronize(stopEvent) );
  GCE( hipEventElapsedTime(&ms, startEvent, stopEvent) );
  printf("Time for asynchronous V1 transfer and execute (ms): %f\n", ms);
  printf("  max error: %e\n", maxError(a, n));
  
  
  // asynchronous version 2: 
  // loop over copy, loop over kernel, loop over copy
  memset(a, 0, bytes);
  GCE( hipEventRecord(startEvent,0) );
  for (int i = 0; i < nStreams; ++i) {
    int offset = i * streamSize;
    GCE( hipMemcpyAsync(&d_a[offset], &a[offset], 
                               streamBytes, hipMemcpyHostToDevice,
                               stream[i]) );
  }
  for (int i = 0; i < nStreams; ++i ){
    int offset = i * streamSize;
    kernel<<<streamSize/blockSize, blockSize, 0, stream[i]>>>(d_a, offset);
  }
  
  for (int i = 0; i < nStreams; ++i) {
    int offset = i * streamSize;
    GCE( hipMemcpyAsync(&a[offset], &d_a[offset], 
                               streamBytes, hipMemcpyDeviceToHost,
                               stream[i]) );
  }
  GCE( hipEventRecord(stopEvent, 0) );
  GCE( hipEventSynchronize(stopEvent) );
  GCE( hipEventElapsedTime(&ms, startEvent, stopEvent) );
  printf("Time for asynchronous V2 transfer and execute (ms): %f\n", ms);
  printf("  max error: %e\n", maxError(a, n));

 GCE( hipEventDestroy(startEvent) );
  GCE( hipEventDestroy(stopEvent) );
  for (int i = 0; i < nStreams; ++i)
    GCE( hipStreamDestroy(stream[i]) );
  hipFree(d_a);
  hipHostFree(a);

  return 0;
}
